
#include <hip/hip_runtime.h>
// Tests that ptxas and fatbinary are invoked correctly during CUDA
// compilation.
//
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// Regular compiles with -O{0,1,2,3,4,fast}.  -O4 and -Ofast map to ptxas O3.
// RUN: %clang -### -target x86_64-linux-gnu -O0 -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM20,OPT0 %s
// RUN: %clang -### -target x86_64-linux-gnu -O1 -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM20,OPT1 %s
// RUN: %clang -### -target x86_64-linux-gnu -O2 -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM20,OPT2 %s
// RUN: %clang -### -target x86_64-linux-gnu -O3 -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM20,OPT3 %s
// RUN: %clang -### -target x86_64-linux-gnu -O4 -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM20,OPT3 %s
// RUN: %clang -### -target x86_64-linux-gnu -Ofast -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM20,OPT3 %s
// Generating relocatable device code
// RUN: %clang -### -target x86_64-linux-gnu -fcuda-rdc -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM20,RDC %s

// With debugging enabled, ptxas should be run with with no ptxas optimizations.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-noopt-device-debug -O2 -g -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM20,DBG %s

// --no-cuda-noopt-device-debug overrides --cuda-noopt-device-debug.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-noopt-device-debug \
// RUN:   --no-cuda-noopt-device-debug -O2 -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM20,OPT2 %s

// Regular compile without -O.  This should result in us passing -O0 to ptxas.
// RUN: %clang -### -target x86_64-linux-gnu -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM20,OPT0 %s

// Regular compiles with -Os and -Oz.  For lack of a better option, we map
// these to ptxas -O3.
// RUN: %clang -### -target x86_64-linux-gnu -Os -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM20,OPT2 %s
// RUN: %clang -### -target x86_64-linux-gnu -Oz -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM20,OPT2 %s

// Regular compile targeting sm_35.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=sm_35 -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35 %s
// Separate compilation targeting sm_35.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=sm_35 -fcuda-rdc -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,RDC %s

// 32-bit compile.
// RUN: %clang -### -target i386-linux-gnu -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH32,SM20 %s
// 32-bit compile when generating relocatable device code.
// RUN: %clang -### -target i386-linux-gnu -fcuda-rdc -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH32,SM20,RDC %s

// Compile with -fintegrated-as.  This should still cause us to invoke ptxas.
// RUN: %clang -### -target x86_64-linux-gnu -fintegrated-as -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM20,OPT0 %s
// Check that we still pass -c when generating relocatable device code.
// RUN: %clang -### -target x86_64-linux-gnu -fintegrated-as -fcuda-rdc -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM20,RDC %s

// Check -Xcuda-ptxas and -Xcuda-fatbinary
// RUN: %clang -### -target x86_64-linux-gnu -c -Xcuda-ptxas -foo1 \
// RUN:   -Xcuda-fatbinary -bar1 -Xcuda-ptxas -foo2 -Xcuda-fatbinary -bar2 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,SM20,PTXAS-EXTRA,FATBINARY-EXTRA %s

// MacOS spot-checks
// RUN: %clang -### -target x86_64-apple-macosx -O0 -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM20,OPT0 %s
// RUN: %clang -### -target x86_64-apple-macosx --cuda-gpu-arch=sm_35 -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35 %s
// RUN: %clang -### -target i386-apple-macosx -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH32,SM20 %s

// Check relocatable device code generation on MacOS.
// RUN: %clang -### -target x86_64-apple-macosx -O0 -fcuda-rdc -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM20,RDC %s
// RUN: %clang -### -target x86_64-apple-macosx --cuda-gpu-arch=sm_35 -fcuda-rdc -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,RDC %s
// RUN: %clang -### -target i386-apple-macosx -fcuda-rdc -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH32,SM20,RDC %s

// Check that CLANG forwards the -v flag to PTXAS.
// RUN:   %clang -### -save-temps -no-canonical-prefixes -v %s 2>&1 \
// RUN:   | FileCheck -check-prefix=CHK-PTXAS-VERBOSE %s

// Match clang job that produces PTX assembly.
// CHECK: "-cc1"
// ARCH64-SAME: "-triple" "nvptx64-nvidia-cuda"
// ARCH32-SAME: "-triple" "nvptx-nvidia-cuda"
// SM20-SAME: "-target-cpu" "sm_20"
// SM35-SAME: "-target-cpu" "sm_35"
// SM20-SAME: "-o" "[[PTXFILE:[^"]*]]"
// SM35-SAME: "-o" "[[PTXFILE:[^"]*]]"
// RDC-SAME: "-fcuda-rdc"
// CHECK-NOT: "-fcuda-rdc"

// Match the call to ptxas (which assembles PTX to SASS).
// CHECK: ptxas
// ARCH64-SAME: "-m64"
// ARCH32-SAME: "-m32"
// OPT0-SAME: "-O0"
// OPT0-NOT: "-g"
// OPT1-SAME: "-O1"
// OPT1-NOT: "-g"
// OPT2-SAME: "-O2"
// OPT2-NOT: "-g"
// OPT3-SAME: "-O3"
// OPT3-NOT: "-g"
// DBG-SAME: "-g" "--dont-merge-basicblocks" "--return-at-end"
// SM20-SAME: "--gpu-name" "sm_20"
// SM35-SAME: "--gpu-name" "sm_35"
// SM20-SAME: "--output-file" "[[CUBINFILE:[^"]*]]"
// SM35-SAME: "--output-file" "[[CUBINFILE:[^"]*]]"
// CHECK-SAME: "[[PTXFILE]]"
// PTXAS-EXTRA-SAME: "-foo1"
// PTXAS-EXTRA-SAME: "-foo2"
// RDC-SAME: "-c"
// CHECK-NOT: "-c"

// Match the call to fatbinary (which combines all our PTX and SASS into one
// blob).
// CHECK: fatbinary
// CHECK-SAME-DAG: "--cuda"
// ARCH64-SAME-DAG: "-64"
// ARCH32-SAME-DAG: "-32"
// CHECK-DAG: "--create" "[[FATBINARY:[^"]*]]"
// SM20-SAME-DAG: "--image=profile=compute_20,file=[[PTXFILE]]"
// SM35-SAME-DAG: "--image=profile=compute_35,file=[[PTXFILE]]"
// SM20-SAME-DAG: "--image=profile=sm_20,file=[[CUBINFILE]]"
// SM35-SAME-DAG: "--image=profile=sm_35,file=[[CUBINFILE]]"
// FATBINARY-EXTRA-SAME: "-bar1"
// FATBINARY-EXTRA-SAME: "-bar2"

// Match the clang job for host compilation.
// CHECK: "-cc1"
// ARCH64-SAME: "-triple" "x86_64-
// ARCH32-SAME: "-triple" "i386-
// CHECK-SAME: "-fcuda-include-gpubinary" "[[FATBINARY]]"
// RDC-SAME: "-fcuda-rdc"
// CHECK-NOT: "-fcuda-rdc"

// CHK-PTXAS-VERBOSE: ptxas{{.*}}" "-v"
